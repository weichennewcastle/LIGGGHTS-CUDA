#include "hip/hip_runtime.h"
/* ----------------------------------------------------------------------
   LAMMPS - Large-scale Atomic/Molecular Massively Parallel Simulator 

   Original Version:
   http://lammps.sandia.gov, Sandia National Laboratories
   Steve Plimpton, sjplimp@sandia.gov 

   See the README file in the top-level LAMMPS directory. 

   ----------------------------------------------------------------------- 

   USER-CUDA Package and associated modifications:
   https://sourceforge.net/projects/lammpscuda/ 

   Christian Trott, christian.trott@tu-ilmenau.de
   Lars Winterfeld, lars.winterfeld@tu-ilmenau.de
   Theoretical Physics II, University of Technology Ilmenau, Germany 

   See the README file in the USER-CUDA directory. 

   This software is distributed under the GNU General Public License.
------------------------------------------------------------------------- */

__global__ void Cuda_AtomVecGranularCuda_PackComm_Kernel(int* sendlist,int n,int maxlistlength,int iswap,X_FLOAT dx,X_FLOAT dy,X_FLOAT dz)
{
  int i=(blockIdx.x*gridDim.y+blockIdx.y)*blockDim.x+threadIdx.x;
  int* list=sendlist+iswap*maxlistlength;
  if(i<n)
  {
    int j=list[i];
  
    ((X_FLOAT*) _buffer)[i]=_x[j] + dx;
    ((X_FLOAT*) _buffer)[i+1*n] = _x[j+_nmax] + dy;
    ((X_FLOAT*) _buffer)[i+2*n] = _x[j+2*_nmax] + dz;
  }
}

__global__ void Cuda_AtomVecGranularCuda_PackCommVel_Kernel(int* sendlist,int n,int maxlistlength,int iswap,X_FLOAT dx,X_FLOAT dy,X_FLOAT dz)
{
  int i=(blockIdx.x*gridDim.y+blockIdx.y)*blockDim.x+threadIdx.x;
  int* list=sendlist+iswap*maxlistlength;
  if(i<n)
  {
    int j=list[i];
  
    ((X_FLOAT*) _buffer)[i]=_x[j] + dx;
    ((X_FLOAT*) _buffer)[i+1*n] = _x[j+_nmax] + dy;
    ((X_FLOAT*) _buffer)[i+2*n] = _x[j+2*_nmax] + dz;
    ((X_FLOAT*) _buffer)[i+3*n]=_v[j];
    ((X_FLOAT*) _buffer)[i+4*n] = _v[j+_nmax];
    ((X_FLOAT*) _buffer)[i+5*n] = _v[j+2*_nmax];
  }
}

__global__ void Cuda_AtomVecGranularCuda_PackComm_Self_Kernel(int* sendlist,int n,int maxlistlength,int iswap,X_FLOAT dx,X_FLOAT dy,X_FLOAT dz,int first)
{
  int i=(blockIdx.x*gridDim.y+blockIdx.y)*blockDim.x+threadIdx.x;
  int* list=sendlist+iswap*maxlistlength;
  if(i<n)
  {
    int j=list[i];
  
    _x[i+first]=_x[j] + dx;
    _x[i+first+_nmax] = _x[j+_nmax] + dy;
    _x[i+first+2*_nmax] = _x[j+2*_nmax] + dz;
  }
  
}

__global__ void Cuda_AtomVecGranularCuda_PackCommVel_Self_Kernel(int* sendlist,int n,int maxlistlength,int iswap,X_FLOAT dx,X_FLOAT dy,X_FLOAT dz,int first)
{
  int i=(blockIdx.x*gridDim.y+blockIdx.y)*blockDim.x+threadIdx.x;
  int* list=sendlist+iswap*maxlistlength;
  if(i<n)
  {
    int j=list[i];
  
    _x[i+first]=_x[j] + dx;
    _x[i+first+_nmax] = _x[j+_nmax] + dy;
    _x[i+first+2*_nmax] = _x[j+2*_nmax] + dz;
    _v[i+first]=_v[j];
    _v[i+first+_nmax] = _v[j+_nmax];
    _v[i+first+2*_nmax] = _v[j+2*_nmax];
  }
  
}

__global__ void Cuda_AtomVecGranularCuda_UnpackComm_Kernel(int n,int first)
{
  int i=(blockIdx.x*gridDim.y+blockIdx.y)*blockDim.x+threadIdx.x;
  if(i<n)
  {
  _x[i+first]=((X_FLOAT*) _buffer)[i];
  _x[i+first+_nmax]=((X_FLOAT*) _buffer)[i+1*n];
  _x[i+first+2*_nmax]=((X_FLOAT*) _buffer)[i+2*n];
  }
}

__global__ void Cuda_AtomVecGranularCuda_UnpackCommVel_Kernel(int n,int first)
{
  int i=(blockIdx.x*gridDim.y+blockIdx.y)*blockDim.x+threadIdx.x;
  if(i<n)
  {
  _x[i+first]=((X_FLOAT*) _buffer)[i];
  _x[i+first+_nmax]=((X_FLOAT*) _buffer)[i+1*n];
  _x[i+first+2*_nmax]=((X_FLOAT*) _buffer)[i+2*n];
  _v[i+first]=((X_FLOAT*) _buffer)[i+3*n];
  _v[i+first+_nmax]=((X_FLOAT*) _buffer)[i+4*n];
  _v[i+first+2*_nmax]=((X_FLOAT*) _buffer)[i+5*n];
  }
}

__global__ void Cuda_AtomVecGranularCuda_PackReverse_Kernel(int n,int first)
{
  int i=(blockIdx.x*gridDim.y+blockIdx.y)*blockDim.x+threadIdx.x;
  if(i<n)
  {
  ((F_FLOAT*) _buffer)[i]=_f[i+first];
  ((F_FLOAT*) _buffer)[i+n] = _f[i+first+_nmax];
  ((F_FLOAT*) _buffer)[i+2*n] = _f[i+first+2*_nmax];
  }
  
}

__global__ void Cuda_AtomVecGranularCuda_UnpackReverse_Kernel(int* sendlist,int n,int maxlistlength,int iswap)
{
  int i=(blockIdx.x*gridDim.y+blockIdx.y)*blockDim.x+threadIdx.x;
  int* list=sendlist+iswap*maxlistlength;
  if(i<n)
  {
  int j=list[i];
  _f[j]+=((F_FLOAT*)_buffer)[i];
  _f[j+_nmax]+=((F_FLOAT*) _buffer)[i+n];
  _f[j+2*_nmax]+=((F_FLOAT*) _buffer)[i+2*n];
  }
  
}

__global__ void Cuda_AtomVecGranularCuda_UnpackReverse_Self_Kernel(int* sendlist,int n,int maxlistlength,int iswap,int first)
{
  int i=(blockIdx.x*gridDim.y+blockIdx.y)*blockDim.x+threadIdx.x;
  int* list=sendlist+iswap*maxlistlength;
  if(i<n)
  {
  int j=list[i];
  
  _f[j]+=_f[i+first];
  _f[j+_nmax]+=_f[i+first+_nmax];
  _f[j+2*_nmax]+=_f[i+first+2*_nmax];
  }
  
}


__global__ void Cuda_AtomVecGranularCuda_PackExchange_Kernel(int n,int dim)
{
  int i=(blockIdx.x*gridDim.y+blockIdx.y)*blockDim.x+threadIdx.x;
  double* buf=(double*) _buffer;
  buf=&buf[1];
  
  if(i<_nlocal)
  {
  	
    if (static_cast <double> (_x[i+dim*_nmax]) < _sublo[dim] || static_cast <double> (_x[i+dim*_nmax]) > _subhi[dim]) //only send if atom is really outside (in original lammps its compared vs >=_subhi
    {
	  int j=atomicAdd((int*)_buffer,1);
	  if(NCUDAEXCHANGE*(j+1)<n)
	  {
	    buf=&buf[NCUDAEXCHANGE*j];
	    buf[0]=i;
	    buf[1]=static_cast <double> (_x[i]);
	    buf[2]=static_cast <double> (_x[i+_nmax]);
	    buf[3]=static_cast <double> (_x[i+2*_nmax]);
	    buf[4]=_v[i];
	    buf[5]=_v[i+_nmax];
	    buf[6]=_v[i+2*_nmax];
	    int atag=_tag[i];
	    buf[7]=atag<0?-atag:atag;
	    _tag[i]=atag<0?atag:-atag;
	    buf[8]=_type[i];
	    buf[9]=_mask[i];
	    buf[10]=_image[i];
	  }
    }
  }
}

__global__ void Cuda_AtomVecGranularCuda_PackExchange_FillExchanges_Kernel()
{
  int i=(blockIdx.x*gridDim.y+blockIdx.y)*blockDim.x+threadIdx.x;

  double* buf=(double*) _buffer;
  int nsend=((int*) _buffer)[0];
  buf=&buf[1];

  int j=0;
  int k=_nlocal;
  int sendlist=-1;
  int replacelist=-1;
  if(i<nsend)
  {
    j=static_cast <int> (buf[NCUDAEXCHANGE*i]);
    if(j<_nlocal-nsend)
    for(int l=0;l<=i;l++)
    if(static_cast <int> (buf[NCUDAEXCHANGE*l])<_nlocal-nsend) sendlist++;
  }
  __syncthreads();
  if((i<nsend)&&(j<_nlocal-nsend))
  {  
    for(int l=_nlocal-1;l>=_nlocal-nsend;l--)
    {
      if(_tag[l]>0) {replacelist++; if(replacelist==sendlist) k=l;}
    }
  }
  
  __syncthreads();
  if((j<_nlocal-nsend)&&(i<nsend))
  {
    _x[j]=_x[k];
    _x[j+_nmax]=_x[k+_nmax];
    _x[j+2*_nmax]=_x[k+2*_nmax];  
    _v[j]=_v[k];
    _v[j+_nmax]=_v[k+_nmax];
    _v[j+2*_nmax]=_v[k+2*_nmax];  
    _tag[j]=_tag[k];
    _type[j]=_type[k];
    _mask[j]=_mask[k];
    _image[j]=_image[k];
  } 
}

__global__ void Cuda_AtomVecGranularCuda_UnpackExchange_Kernel(int dim)
{
  int i=(blockIdx.x*gridDim.y+blockIdx.y)*blockDim.x+threadIdx.x;
  double* buf=(double*) _buffer;
  buf=&buf[1];
  int n_add=static_cast<int> (((double*)_buffer)[0]);
  
  if(i<n_add)
  {
	    buf=&buf[NCUDAEXCHANGE*i];
 	    if(buf[1+dim]>=_sublo[dim] && buf[1+dim]<_subhi[dim])
	   	{
	   	  int j=atomicAdd(_flag,1)+_nlocal;
	   	
	      _x[j]=buf[1];
	      _x[j+_nmax]=buf[2];
	      _x[j+2*_nmax]=buf[3];
	      _v[j]=buf[4];
	      _v[j+_nmax]=buf[5];
	      _v[j+2*_nmax]=buf[6];
	      _tag[j]=buf[7];
	      _type[j]=buf[8];
	      _mask[j]=buf[9];
	      _image[j]=buf[10];  
	   	}
  }
}

__global__ void Cuda_AtomVecGranularCuda_PackBorder_Kernel(int* sendlist,int n,int maxlistlength,int iswap,X_FLOAT dx,X_FLOAT dy,X_FLOAT dz)
{
  int i=(blockIdx.x*gridDim.y+blockIdx.y)*blockDim.x+threadIdx.x;
  int* list=sendlist+iswap*maxlistlength;
  if(i<n)
  {
    int j=list[i];
  
    ((X_FLOAT*) _buffer)[i]=_x[j] + dx;
    ((X_FLOAT*) _buffer)[i+1*n] = _x[j+_nmax] + dy;
    ((X_FLOAT*) _buffer)[i+2*n] = _x[j+2*_nmax] + dz;
    ((X_FLOAT*) _buffer)[i+3*n] = _tag[j];
    ((X_FLOAT*) _buffer)[i+4*n] = _type[j];
    ((X_FLOAT*) _buffer)[i+5*n] = _mask[j];
  }
  
}

__global__ void Cuda_AtomVecGranularCuda_PackBorderVel_Kernel(int* sendlist,int n,int maxlistlength,int iswap,X_FLOAT dx,X_FLOAT dy,X_FLOAT dz)
{
  int i=(blockIdx.x*gridDim.y+blockIdx.y)*blockDim.x+threadIdx.x;
  int* list=sendlist+iswap*maxlistlength;
  if(i<n)
  {
    int j=list[i];
  
    ((X_FLOAT*) _buffer)[i]=_x[j] + dx;
    ((X_FLOAT*) _buffer)[i+1*n] = _x[j+_nmax] + dy;
    ((X_FLOAT*) _buffer)[i+2*n] = _x[j+2*_nmax] + dz;
    ((X_FLOAT*) _buffer)[i+3*n] = _tag[j];
    ((X_FLOAT*) _buffer)[i+4*n] = _type[j];
    ((X_FLOAT*) _buffer)[i+5*n] = _mask[j];
    ((X_FLOAT*) _buffer)[i+6*n] = _v[j];
    ((X_FLOAT*) _buffer)[i+7*n] = _v[j+_nmax];
    ((X_FLOAT*) _buffer)[i+8*n] = _v[j+2*_nmax];
  }
  
}

__global__ void Cuda_AtomVecGranularCuda_PackBorder_Self_Kernel(int* sendlist,int n,int maxlistlength,int iswap,X_FLOAT dx,X_FLOAT dy,X_FLOAT dz,int first)
{
  int i=(blockIdx.x*gridDim.y+blockIdx.y)*blockDim.x+threadIdx.x;
  int* list=sendlist+iswap*maxlistlength;
  if(i<n)
  {
    int j=list[i];
  
    _x[i+first]=_x[j] + dx;
    _x[i+first+_nmax] = _x[j+_nmax] + dy;
    _x[i+first+2*_nmax] = _x[j+2*_nmax] + dz;
	_tag[i+first] = _tag[j];
	_type[i+first] = _type[j];
	_mask[i+first] = _mask[j];
  }
}

__global__ void Cuda_AtomVecGranularCuda_UnpackBorder_Kernel(int n,int first)
{
  int i=(blockIdx.x*gridDim.y+blockIdx.y)*blockDim.x+threadIdx.x;
  if(i<n)
  {
  	if(i+first<_nmax)
  	{
  	  _x[i+first]=((X_FLOAT*) _buffer)[i];
  	  _x[i+first+_nmax]=((X_FLOAT*) _buffer)[i+1*n];
  	  _x[i+first+2*_nmax]=((X_FLOAT*) _buffer)[i+2*n];
  	  _tag[i+first] = static_cast<int> (((X_FLOAT*) _buffer)[i+3*n]);
  	  _type[i+first] = static_cast<int> (((X_FLOAT*) _buffer)[i+4*n]);
  	  _mask[i+first] = static_cast<int> (((X_FLOAT*) _buffer)[i+5*n]);
  	}
  	else
  	{
  	  _flag[0]=1;
  	}
  }
}

__global__ void Cuda_AtomVecGranularCuda_UnpackBorderVel_Kernel(int n,int first)
{
  int i=(blockIdx.x*gridDim.y+blockIdx.y)*blockDim.x+threadIdx.x;
  if(i<n)
  {
  	if(i+first<_nmax)
  	{
  	  _x[i+first]=((X_FLOAT*) _buffer)[i];
  	  _x[i+first+_nmax]=((X_FLOAT*) _buffer)[i+1*n];
  	  _x[i+first+2*_nmax]=((X_FLOAT*) _buffer)[i+2*n];
  	  _tag[i+first] = static_cast<int> (((X_FLOAT*) _buffer)[i+3*n]);
  	  _type[i+first] = static_cast<int> (((X_FLOAT*) _buffer)[i+4*n]);
  	  _mask[i+first] = static_cast<int> (((X_FLOAT*) _buffer)[i+5*n]);
  	  _v[i+first]=((X_FLOAT*) _buffer)[i+6*n];
  	  _v[i+first+_nmax]=((X_FLOAT*) _buffer)[i+7*n];
  	  _v[i+first+2*_nmax]=((X_FLOAT*) _buffer)[i+8*n];
  	}
  	else
  	{
  	  _flag[0]=1;
  	}
  }
}
