#include "hip/hip_runtime.h"
/* ----------------------------------------------------------------------
   LAMMPS - Large-scale Atomic/Molecular Massively Parallel Simulator 

   Original Version:
   http://lammps.sandia.gov, Sandia National Laboratories
   Steve Plimpton, sjplimp@sandia.gov 

   See the README file in the top-level LAMMPS directory. 

   ----------------------------------------------------------------------- 

   USER-CUDA Package and associated modifications:
   https://sourceforge.net/projects/lammpscuda/ 

   Christian Trott, christian.trott@tu-ilmenau.de
   Lars Winterfeld, lars.winterfeld@tu-ilmenau.de
   Theoretical Physics II, University of Technology Ilmenau, Germany 

   See the README file in the USER-CUDA directory. 

   This software is distributed under the GNU General Public License.
------------------------------------------------------------------------- */
#define INERTIA 0.4
static inline __device__ void check_distance(X_FLOAT &xtmp,X_FLOAT &ytmp,X_FLOAT &ztmp,int &i,int groupbit)
{
  if(_dist_check)
  {
  	X_FLOAT tmp=xtmp-_xhold[i];
  	X_FLOAT d=tmp*tmp;
  	tmp=ytmp-_xhold[i+_maxhold];
  	d+=tmp*tmp;
  	tmp=ztmp-_xhold[i+2*_maxhold];
  	d+=tmp*tmp;
  
  	d=((i < _nlocal) && (_mask[i] & groupbit))?d:X_F(0.0);
  
  	if(not __all(d<=_triggerneighsq)) 
  		_reneigh_flag[0]=1;
  }
}


__global__ void FixNVESphereCuda_InitialIntegrate_Kernel(int groupbit)
{
	X_FLOAT xtmp,ytmp,ztmp;
	
	int i=(blockIdx.x*gridDim.y+blockIdx.y)*blockDim.x+threadIdx.x;
	if(i < _nlocal && _mask[i] & groupbit)
	{
  		V_FLOAT dtfrotate = _dtf / INERTIA;
		F_FLOAT* my_f = _f + i;
		V_FLOAT* my_v = _v + i;
		X_FLOAT* my_x = _x + i;
				
	  	V_FLOAT dtirotate = _radius[i];

	    V_FLOAT			mass = _rmass_flag?_rmass[i]:_mass[_type[i]];
		V_FLOAT 		dtfm = _dtf * (V_F(1.0) / mass);

	  	dtirotate*=dtirotate*mass;

		V_FLOAT v_mem;
		v_mem = *my_v += dtfm * (*my_f); xtmp=*my_x += _dtv * v_mem;  my_f += _nmax; my_v += _nmax; my_x += _nmax;
		v_mem = *my_v += dtfm * (*my_f); ytmp=*my_x += _dtv * v_mem;  my_f += _nmax; my_v += _nmax; my_x += _nmax;
		v_mem = *my_v += dtfm * (*my_f); ztmp=*my_x += _dtv * v_mem;
	  	
	  	dtirotate=(V_F(1.0)/dtirotate)*dtfrotate;
	  	_omega[i] += dtirotate * _torque[i];
	  	_omega[i+_nmax] += dtirotate * _torque[i+_nmax];
	  	_omega[i+2*_nmax] += dtirotate * _torque[i+2*_nmax];
	}
	
	check_distance(xtmp,ytmp,ztmp,i,groupbit);
}

__global__ void FixNVESphereCuda_FinalIntegrate_Kernel(int groupbit)
{
	
	int i=(blockIdx.x*gridDim.y+blockIdx.y)*blockDim.x+threadIdx.x;
	if(i < _nlocal && _mask[i] & groupbit)
	{
  		V_FLOAT dtfrotate = _dtf / INERTIA;
		F_FLOAT* my_f = _f + i;
		V_FLOAT* my_v = _v + i;

	  	V_FLOAT dtirotate = _radius[i];

	    V_FLOAT			mass = _rmass_flag?_rmass[i]:_mass[_type[i]];
		V_FLOAT 		dtfm = _dtf * (V_F(1.0) / mass);

	  	dtirotate*=dtirotate*mass;

		*my_v += dtfm * (*my_f);  my_f += _nmax; my_v += _nmax;
		*my_v += dtfm * (*my_f);  my_f += _nmax; my_v += _nmax;
		*my_v += dtfm * (*my_f);

	  	dtirotate=(V_F(1.0)/dtirotate)*dtfrotate;
	  	_omega[i] += dtirotate * _torque[i];
	  	_omega[i+_nmax] += dtirotate * _torque[i+_nmax];
	  	_omega[i+2*_nmax] += dtirotate * _torque[i+2*_nmax];
	}
	
}



